#include "hip/hip_runtime.h"
#include <stdio.h>
#include <glew/glew.h>
#include <GL/freeglut.h>
#include <GL/GL.h>
#include <GL/GLU.h>
#include <GL/glut.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"

const int WIDTH = 1920;
const int HEIGHT = 1080;
const int COUNT = 30000000;

//float4* positions;
float2* velocity;
float2* xy;

__global__ void simulateFrame(float* positions, float2* velocity, float2* xy, float delta, float particleRadius)
{
	delta /= 1000.0;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= COUNT) return;

	xy[idx].x += velocity[idx].x * delta;
	xy[idx].y += velocity[idx].y * delta;
	if (xy[idx].x < 0 + particleRadius) {
		velocity[idx].x *= -1;
		xy[idx].x *= -1;
	}
	if (xy[idx].x + particleRadius >= WIDTH) {
		velocity[idx].x *= -1;
		xy[idx].x -= 2 * ((xy[idx].x + particleRadius) - WIDTH);
	}
	if (xy[idx].y < 0 + particleRadius) {
		velocity[idx].y *= -1;
		xy[idx].y *= -1;
	}
	if (xy[idx].y + particleRadius >= HEIGHT) {
		velocity[idx].y *= -1;
		xy[idx].y -= 2 * ((xy[idx].y + particleRadius) - HEIGHT);
	}
	positions[idx * 3] = xy[idx].x;
	positions[idx * 3 + 1] = xy[idx].y;

}

__global__ void setVal(float2* arr, float2 val) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= COUNT) return;
	arr[idx] = val;
}

void initFrame();

void displayFrame();

GLuint positionsVBO;
struct hipGraphicsResource* positionsVBO_CUDA;

int main(int argv, char ** argc)
{
	float2* h_velocity = new float2[COUNT];

	for (int idx = 0; idx < COUNT; idx++)
	{
		h_velocity[idx].x = rand() % 200 + 10;
		if (rand() % 2)
			h_velocity[idx].x *= -1;
		h_velocity[idx].y = rand() % 200 + 10;
		if (rand() % 2)
			h_velocity[idx].y *= -1;
	}

	hipMalloc(&velocity, COUNT * sizeof(float2));
	hipMemcpy(velocity, h_velocity, COUNT * sizeof(float2), hipMemcpyHostToDevice);
	hipMalloc(&xy, COUNT * sizeof(float2));
	setVal<<<(COUNT+1023)/1024, 1024>>>(xy, make_float2(100, 100));
	//cudaGLSetGLDevice(0);

	glutInit(&argv, argc);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);

	glutInitWindowPosition(0, 0);
	glutInitWindowSize(WIDTH, HEIGHT);
	glutCreateWindow("Particles");

	glewInit();

	initFrame();
	glutDisplayFunc(displayFrame);



	glutMainLoop();
	return 0;
}

GLfloat* positions;

int lastTime = 0;
void initFrame(){
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glPointSize(4.0);
	glLineWidth(2.0);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, WIDTH, 0.0, HEIGHT);
	glGenBuffers(1, &positionsVBO);
	glBindBuffer(GL_ARRAY_BUFFER, positionsVBO);
	unsigned int size = WIDTH * HEIGHT * 4 * sizeof(GLfloat);
	glBufferData(GL_ARRAY_BUFFER, size, positions, GL_DYNAMIC_DRAW);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);
	hipGraphicsGLRegisterBuffer(&positionsVBO_CUDA, positionsVBO, cudaGraphicsMapFlagsWriteDiscard);
	lastTime = glutGet(GLUT_ELAPSED_TIME);
}

//void displayFrame() {
//	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
//	glColor3f(1.0, 0.0, 1.0);
//
//	hipGraphicsMapResources(1, &positionsVBO_CUDA, 0);
//	size_t num_bytes;
//	hipGraphicsResourceGetMappedPointer((void**)&positions, &num_bytes, positionsVBO_CUDA);
//	// Execute kernel
//	simulateFrame <<<(COUNT + 1023) / 1024, 1024 >>>(positions, velocity, xy, glutGet(GLUT_ELAPSED_TIME), 1.0);
//	// Unmap buffer object
//	hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
//	// Render from buffer object
//	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
//	glBindBuffer(GL_ARRAY_BUFFER, positionsVBO);
//	glVertexPointer(4, GL_FLOAT, 0, 0);
//	glEnableClientState(GL_VERTEX_ARRAY);
//	glDrawArrays(GL_POINTS, 0, WIDTH * HEIGHT);
//	glDisableClientState(GL_VERTEX_ARRAY);
//	// Swap buffers
//	glutSwapBuffers();
//	glutPostRedisplay();
//	/*
//	glBegin(GL_POINTS);
//	glVertex2f(10.0, 10.0);
//	glVertex2f(10.0, 30.0);
//	glEnd();
//	glFlush();
//	*/
//}
void displayFrame() {
	int ms = glutGet(GLUT_ELAPSED_TIME) - lastTime;
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glColor3f(1.0, 0.0, 1.0);

	hipGraphicsMapResources(1, &positionsVBO_CUDA, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void**)&positions, &num_bytes, positionsVBO_CUDA);
	// Execute kernel
	lastTime = glutGet(GLUT_ELAPSED_TIME);
	simulateFrame <<<(COUNT + 1023) / 1024, 1024 >>>(positions, velocity, xy, ms, 4.0);
	// Unmap buffer object
	hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
	// Render from buffer object
	glBindBuffer(GL_ARRAY_BUFFER, positionsVBO);
	glVertexPointer(3, GL_FLOAT, 0, 0);
	glEnableClientState(GL_VERTEX_ARRAY);
	glColor3f(1.0, 1.0, 1.0);
	glDrawArrays(GL_POINTS, 0, COUNT);
	glDisableClientState(GL_VERTEX_ARRAY);
	// Swap buffers
	glutSwapBuffers();
	glutPostRedisplay();
	/*
	glBegin(GL_POINTS);
	glVertex2f(10.0, 10.0);
	glVertex2f(10.0, 30.0);
	glEnd();
	glFlush();
	*/
}